#include "hip/hip_runtime.h"
#include "device_mine.cuh"
#include "../util/utils.hpp"
#include <iomanip>
#include <sstream>

using namespace std;

uint32_t CPU_mine(string payload, uint32_t difficulty) {
    for(uint32_t nonce = 0; nonce<0xffffffff; nonce++) {
        stringstream ss;
        ss << payload << hex << nonce;
        string hash = hash_sha256(ss.str());
        if (hash.substr(0, difficulty) == string(difficulty, '0')) {
            return nonce;
        }
    }
    return 0;
}

__global__ void GPU_naive_mine(string payload, uint32_t difficulty, uint32_t* nonce, bool* nonce_found) {
    if (!nonce_found[0]){
        uint32_t candidate_nonce = gridDim.x*blockDim.x*blockIdx.y + blockDim.x*blockIdx.x + threadIdx.x;
        stringstream ss;
        ss << payload << hex << candidate_nonce;
        string hash = hash_sha256(ss.str());
        if (hash.substr(0, difficulty) == string(difficulty, '0')){
            nonce[0] = candidate_nonce;
            nonce_found[0] = true;
        }
    }
}



uint32_t device_mine_dispatcher(string payload, uint32_t difficulty, MineType reduction_type) {
    switch (reduction_type) {
        case MineType::MINE_CPU: {
            return CPU_mine(payload, difficulty);
        }
        case MineType::MINE_NAIVE: {
            uint32_t *nonce;
            bool *nonce_found;
            hipMallocManaged(&nonce, 2*sizeof(uint32_t));
            hipMallocManaged(&nonce_found, 2*sizeof(bool));
            nonce_found[0] = false;
            int blockSize = 1024;
            int numBlocks = 4194304; // ceil(0xffffffff/1024)
            GPU_naive_mine<<1, 1024>>(payload, difficulty, nonce, nonce_found);
            hipDeviceSynchronize();
            return nonce[0];
        }
    }
    return 0;
}